#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100000000

__global__ void dot_kernel(float *a, float *b, float *c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    float temp = 0.0f;
    for (int j = 0; j < N; j += blockDim.x * gridDim.x) {
        temp += a[i + j] * b[i + j];
    }
    c[i] = temp;
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    h_a = (float*)malloc(N * sizeof(float));
    h_b = (float*)malloc(N * sizeof(float));
    h_c = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        h_a[i] = i * 2.0f;
        h_b[i] = i * 3.0f;
    }
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    int block_size = 256;
    int num_blocks = (N + block_size - 1) / block_size;
    dot_kernel<<<num_blocks, block_size>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
    float dot = 0.0f;
    for (int i = 0; i < N; i++) {
        dot += h_c[i];
    }
    printf("Dot product: %f\n", dot);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}